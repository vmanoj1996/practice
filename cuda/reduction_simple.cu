#include <hip/hip_runtime.h>
#include<iostream>

// inputs are device pointers
__global__ void reduce_sum(const float* input, float* output, int N)
{   
    int gid = threadIdx.x + blockDim.x * blockIdx.x;

    extern __shared__ float sdata[];

    // local index
    int tid = threadIdx.x;

    // load the data into shared memory
    sdata[tid] = (gid<N)? input[gid] : 0.0f;
    __syncthreads();

    // reduction loop
    for(int layer_size = blockDim.x/2; layer_size>0; layer_size/=2)
    {
        if(tid<layer_size)
        {
            sdata[tid] += sdata[tid + layer_size];
        }
        __syncthreads();
    }

    if(tid == 0)
    {
        output[blockIdx.x] = sdata[0];
    }
    
}


int main()
{
    float *a, *b;
    const int N = 256;

    hipMallocManaged(&a, N*sizeof(int));
    hipMallocManaged(&b, N*sizeof(int));

    for(int i=0; i<N; i++)
    {
        a[i] = 1;
    }

    int threadsize = 256;
    int blockCount = (threadsize + N - 1)/threadsize;

    reduce_sum<<<blockCount, threadsize, threadsize*sizeof(float)>>>(a, b, N);

    hipDeviceSynchronize();

    std::cout<<b[0]<<std::endl;

    hipFree(a);
    hipFree(b);

    return 0;
}